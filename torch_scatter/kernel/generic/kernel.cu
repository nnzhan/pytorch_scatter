
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

void check(THCState *state, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, output, input));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 2, index));
  THArgCheck(THCTensor_(nDimension)(state, output) <= MAX_DIMS, 1, "Tensor too large or too many dimensions");
}

void scatter_(mul)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  check(state, output, index, input);

  const ptrdiff_t n = THCudaLongTensor_nElement(state, index);
  const dim3 block = dim3(NUM_THREADS);
  printf("mul");
}

void scatter_(div)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  check(state, output, index, input);
  printf("div");
}

void scatter_(mean)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCTensor *num_output) {
  check(state, output, index, input);
  printf("mean");
}

void scatter_(max)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg_output) {
  check(state, output, index, input);
  printf("max");
}

void scatter_(min)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg_output) {
  check(state, output, index, input);
  printf("min");
}

void index_backward(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *grad, THCudaLongTensor *arg_grad) {
  check(state, output, index, grad);
  printf("index_backward");
}

#endif